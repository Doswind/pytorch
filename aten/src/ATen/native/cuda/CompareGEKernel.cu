#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>


// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {
template<typename scalar_t>
struct CompareFunctor{
  CompareFunctor(const int op): op_(op) {TORCH_INTERNAL_ASSERT_DEBUG_ONLY(op_>=0 && op_ <= 3);}
  const int op_;
  __device__ __forceinline__ bool operator() (scalar_t a, scalar_t b) const {
    //printf("vals %ld %ld\n", a, b);
    if (op_ == 0) {
      return a >= b;
    } else if (op_ == 1) {
      return a > b;
    } else if (op_ == 2) {
      return a <= b;
    } else if (op_ == 3) { //LT
      return a < b;
    }
  }
};


void ge_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_ALL_TYPES_AND3(kHalf, kBFloat16, kBool, iter.common_dtype(), "ge_cuda", [&]() {
    gpu_kernel_with_scalars(iter, CompareFunctor<scalar_t>(0));
  });
}

void gt_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_ALL_TYPES_AND3(kHalf, kBFloat16, kBool, iter.common_dtype(), "ge_cuda", [&]() {
    gpu_kernel_with_scalars(iter, CompareFunctor<scalar_t>(1));
  });
}

void le_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_ALL_TYPES_AND3(kHalf, kBFloat16, kBool, iter.common_dtype(), "ge_cuda", [&]() {
    gpu_kernel_with_scalars(iter, CompareFunctor<scalar_t>(2));
  });
}

void lt_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_ALL_TYPES_AND3(kHalf, kBFloat16, kBool, iter.common_dtype(), "ge_cuda", [&]() {
    gpu_kernel_with_scalars(iter, CompareFunctor<scalar_t>(3));
  });
}

REGISTER_DISPATCH(ge_stub, &ge_kernel_cuda);
REGISTER_DISPATCH(gt_stub, &gt_kernel_cuda);
REGISTER_DISPATCH(le_stub, &le_kernel_cuda);
REGISTER_DISPATCH(lt_stub, &lt_kernel_cuda);


// template<typename scalar_t>
// struct CompareGEFunctor {
//   __device__ __forceinline__ bool operator() (scalar_t a, scalar_t b) const {
//     return a >= b;
//   }
// };

// void ge_kernel_cuda(TensorIteratorBase& iter) {
//   AT_DISPATCH_ALL_TYPES_AND3(kHalf, kBFloat16, kBool, iter.common_dtype(), "ge_cuda", [&]() {
//     gpu_kernel_with_scalars(iter, CompareGEFunctor<scalar_t>());
//   });
// }

// REGISTER_DISPATCH(ge_stub, &ge_kernel_cuda);

}} // namespace at::native
